#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define DEBUG 0

#define N 250

int matrixA[N][N], matrixB[N][N], matrixC[N][N], matrixD[N][N];

__global__ void mult_matrix(int matrixA[N][N], int matrixB[N][N], int matrixC[N][N]) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
    int k;

	if (i < N && j < N) {
        for (k = 0; k < N; k++) {
            matrixC[i][j] += matrixA[i][k] * matrixB[k][j];
        }
    }
}

int main() {
	int (*deviceA)[N];
	int (*deviceB)[N];
	int (*deviceC)[N];
	int i, j, k;
    
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			matrixA[i][j] = rand() % 100;
			matrixB[i][j] = rand() % 100;
            matrixC[i][j] = 0;
            matrixD[i][j] = 0;
		}
	}
	
	hipEvent_t start_time, stop_time;
	float elapsedTime;

	hipMalloc((void **) &deviceA, N * N * sizeof(int));
	hipMalloc((void **) &deviceB, N * N * sizeof(int));
	hipMalloc((void **) &deviceC, N * N * sizeof(int));

	hipMemcpy(deviceA, matrixA, N * N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceB, matrixB, N * N * sizeof(int), hipMemcpyHostToDevice);

	dim3 threadsPerBlock(32, 32);
	dim3 numOfBlocks(ceil(N / 32.0), ceil(N / 32.0));
	hipEventCreate(&start_time);
	hipEventRecord(start_time, 0);

	mult_matrix<<<numOfBlocks, threadsPerBlock>>>(deviceA, deviceB, deviceC);
	
	hipEventCreate(&stop_time);
	hipEventRecord(stop_time, 0);
	hipEventSynchronize(stop_time);

	hipEventElapsedTime(&elapsedTime, start_time, stop_time);
	hipMemcpy(matrixC, deviceC, N * N * sizeof(int), hipMemcpyDeviceToHost);
    
#if DEBUG
    printf("\nmatrixA-\n");
    for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
                printf("%d\t", matrixA[i][j]);
		}
		printf("\n");
	}
	
    printf("\nmatrixB-\n");
    for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
                printf("%d\t", matrixB[i][j]);
		}
		printf("\n");
	}
    
    printf("\nmatrixC-\n");
    for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
                printf("%d\t", matrixC[i][j]);
		}
		printf("\n");
	}
	printf("\n");
#endif
	
	printf("Parallely Elapsed Time: %f ms\n", elapsedTime);
	
	clock_t start_time_nonparallely, stop_time_nonparallely;
	start_time_nonparallely = clock();
    
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
            for (k = 0; k < N; k++) {
                matrixD[i][j] += matrixA[i][k] * matrixB[k][j];
            }
		}
	}
	
	stop_time_nonparallely = clock();
	printf("Non-parallely Elapsed Time: %f ms\n", (float) ((stop_time_nonparallely) - (start_time_nonparallely)));
	
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);
    
    return 0;
}
